#include "hip/hip_runtime.h"
#include <c10/cuda/CUDAException.h>
#include <hip/hip_runtime.h>
#include <torch/csrc/distributed/c10d/symm_mem/CUDASymmetricMemory-inl.h>
#include <torch/csrc/distributed/c10d/cuda/Baton.cuh>

namespace c10d::cuda::detail {

__global__
// set launch bounds to limit to 1 thread per block, 1 block per MP, 1 block per
// cluster
__launch_bounds__(1, 1, 1) void kernel_barrier(
    int32_t* value,
    size_t timeout_ms) {
  value[1] = BatonStatus::RUNNING;

  size_t start = c10d::symmetric_memory::global_timer_ns();
  size_t timeout_ns = timeout_ms * 1e6; // Convert milliseconds to nanoseconds
  while (true) {
    // Atomically read the value
    int current_value = atomicAdd(&value[0], 0);
    // Check if the value is equal to the expected value
    if (current_value == 1) {
      value[1] = BatonStatus::ABORTED;
      return;
    }

    if (timeout_ms > 0) {
      // Check if timeout has been reached
      size_t now = c10d::symmetric_memory::global_timer_ns();
      if ((now - start) > timeout_ns) {
        value[1] = BatonStatus::TIMED_OUT;
        return;
      }
    }

    // sleep for 1ms
    __nanosleep(1000000);
  }
}

Baton::Baton(std::chrono::milliseconds timeout)
    : comm_{at::empty({2}, at::TensorOptions().dtype(at::kInt)).pin_memory()},
      timeout_{timeout} {
  // grid size 1, block size 1, 0 bytes of shared memory
  kernel_barrier<<<1, 1, 0>>>(
      comm_.mutable_data_ptr<int32_t>(), timeout_.count());
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

C10_REGISTER_CLASS(BatonRegistry, CUDA, Baton)

} // namespace c10d::cuda::detail
